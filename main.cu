#include "hip/hip_runtime.h"
#include "lib/nlm_lib.h"
#include <chrono>
#include <iostream>
#include "lib/nlm_cu_lib.cuh"

void nlm(std::string& path, std::string& format, std::string& variable,
		 int patchSize, bool noise);

int main(int argc, char** argv) {
  if (argc > 1) {
	bool noise = false;
	std::vector<std::string> paths{};
	std::vector<std::string> formats{};
	std::vector<std::string> variables{};
	std::vector<int> patchSizes{};
	parse_cli_args(argc, argv, paths, formats, variables, patchSizes, noise);
	nlm(paths.at(0), formats.at(0), variables.at(0), patchSizes.at(0), false);
	std::vector<std::vector<double>> image = {
		{1, 2, 3, 4}, {5, 6, 7, 8}, {9, 10, 11, 12}};
	write_csv("file", image);
	return 0;
  } else {
	std::cout << "Wrong usage" << std::endl;
	std::cout << "Try running " << argv[0] << " --help" << std::endl;
  }
  return 0;
}


void nlm(std::string& path, std::string& format, std::string& variable,
		 int patchSize, bool add_noise) {
  std::vector<double> image{};
#ifdef USE_OCTAVE
  if (format == "mat")
	octave_read(path.c_str(), image, variable);
  else {
	std::cerr << "Make sure the format matches your build parameters";
	return;
  }
#elif defined(USE_MATLAB)
  if (format == "mat")
	matread(path.c_str(), image, variable);
  else {
	std::cerr << "Make sure the format matches your build parameters";
	return;
  }
#elif defined(USE_OPENCV)
  if (format == "png")
	opencv_read(path.c_str(), image);
  else {
	std::cerr << "Make sure the format matches your build parameters";
	return;
  }
#else
  int rowCount = 0, columnCount = 0;
  if (format == "csv")
	read_csv(path.c_str(), image, rowCount, columnCount);
  else {
	std::cerr << "Make sure the format matches your build parameters";
	return;
  }
#endif
  // create gaussian noise
  std::vector<double> noise = std::vector<double>(image.size());
  add_gaussian_noise(image, noise.size(), 0.5, noise);
  // create gaussian kernel
  std::vector<std::vector<double>> kernel =
	  std::vector<std::vector<double>>(patchSize);
  for (std::vector<double>& vec : kernel) {
	vec = std::vector<double>(patchSize);
  }
  gaussian_kernel(kernel, patchSize, 0.8);
  // convert 2d vector to 1d
  std::vector<double> kernel1d = std::vector<double>(patchSize * patchSize);
  int i = 0;
  for (std::vector<double>& item : kernel) {
	std::copy(item.begin(), item.end(), kernel1d.begin() + (i * patchSize));
	++i;
  }
  // create necessary vectors
  std::vector<double> denoised_image = std::vector<double>(image.size());
  std::vector<double> host_patch =
	  std::vector<double>(patchSize * patchSize * image.size());
  // declare device variables
  auto* final_image = (double*)malloc(image.size() * sizeof(double));
  auto* h_patch = (double*)malloc(image.size()*kernel1d.size() * sizeof(double));
  double *device_noise, *device_kernel, *patch, *device_denoised;
  // reserve space on device
  hipMalloc((void**)&device_noise, image.size() * sizeof(double));
  hipMalloc((void**)&device_denoised, image.size() * sizeof(double));
  hipMalloc((void**)&device_kernel, kernel1d.size() * sizeof(double));
  hipMalloc((void**)&patch,
			 kernel1d.size() * image.size() * sizeof(double));
  // copy variable data from host to device
  hipMemcpy(device_noise, noise.data(), noise.size()*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_kernel, kernel1d.data(),
			 kernel1d.size() * sizeof(double), hipMemcpyHostToDevice);
  // configure cuda parameters and run on device
  dim3 blockSize(1024);
  dim3 gridSize(1024/ blockSize.x);
  auto start = std::chrono::high_resolution_clock::now();
  int size = (int)std::sqrt(image.size()) +1;
  int ptSize = (int) std::sqrt(kernel1d.size()) +1;
  nlm_cu::findPatches<<<gridSize, blockSize>>>(device_noise, patch, size, patchSize);
  nlm_cu::non_local_means<<<gridSize, blockSize, size>>>(device_noise, image.size(), 0.02, patch, kernel1d.size(),
							  device_kernel, device_denoised);
  hipDeviceSynchronize();
  auto stop = std::chrono::high_resolution_clock::now();
  // copy data from device back to host
  hipMemcpy(final_image, device_denoised, denoised_image.size() * sizeof(double),
			 hipMemcpyDeviceToHost);
  hipMemcpy(h_patch, patch,  kernel1d.size()* image.size()* sizeof(double),
			 hipMemcpyDeviceToHost);
  host_patch = std::vector<double>(h_patch, h_patch+kernel1d.size()*image.size());
  denoised_image = std::vector<double>(final_image, final_image + image.size());
  // convert 1d image vector back to 2d
  std::vector<std::vector<double>> final =
	  std::vector<std::vector<double>>(size);
  i = 0;
  for (std::vector<double>& item : final) {
	item = std::vector<double>(size);
	std::copy(denoised_image.begin() + (i * size),
			  denoised_image.begin() + ((i + 1) * size), item.begin());
	i++;
  }
  // write result to file
  write_csv(path.c_str(), final);
  std::cout << "Took " << (stop - start).count() * 10e-9 << "s" << std::endl;
  // free memory
  hipFree(patch);
  hipFree(device_kernel);
  hipFree(device_denoised);
  hipFree(device_noise);
  free(final_image);
  free(h_patch);
}
